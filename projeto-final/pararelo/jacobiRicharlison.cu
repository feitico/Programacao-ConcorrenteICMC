
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 7

#define check(X)\
{\
    hipError_t cerr = X;\
    if (cerr != hipSuccess){\
        fprintf(stderr, "GPUassert:%s at line%d.\n", hipGetErrorString(cerr), __LINE__);\
        abort();\ 
    }\ 
}\

__global__ 
void add_arrays(int *a) 
{
	a[threadIdx.x] = 21;
}
 
int main()
{
	// Setup the arrays
	int a[N] = {15, 10, 6, 0, -11, 1,0};
  
	int *ad;
	const int isize = N*sizeof(int);
 
	// print the contents of a[]
	for(int i = 0; i < N; i++)
        printf("%d ", a[i]);
 
	// Allocate and Transfer memory to the device
	hipMalloc( (void**)&ad, isize );  
	
	check(hipMemcpy( ad, a, isize, hipMemcpyHostToDevice )); 
	
	// Perform the array addition
	dim3 dimBlock( N  );  
	dim3 dimGrid ( 1  );
	add_arrays<<<dimGrid, dimBlock>>>(ad);
	
	// Copy the Contents from the GPU
	check(hipMemcpy( a, ad, isize, hipMemcpyDeviceToHost )); 
	hipFree( ad );
	
	// print the contents of a[]
	for(int i = 0; i < N; i++)
        printf("%d ", a[i]);
 

	return EXIT_SUCCESS;
}
