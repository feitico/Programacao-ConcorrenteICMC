#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "cump.h"


#define N   100
#define TPB 128 /* indica o no. de threads por bloco */

__global__ void add( int *a, int *b, int *c ) {
    /**
     * threadIdx.x contém o Id da thread a ser executada
     * blockIdx.x contém o Id do bloco
     * blockDim.x cte. que contém o no. de threads utilizadas em cada bloco
     * gridDim.x cte que contém o numero de blocos utilizados em um grid
     */
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // this thread handles the data at its thread id

	if(tid<N){
	        c[tid] = a[tid] + b[tid];
	}
    /**
     * O While adapta a funcao para percorrer vetor maior do que o alocado
     */

//    while (tid < N){
//        c[tid] = a[tid] + b[tid];
//    	tid+= blockDim.x * gridDim.x;
//    }
}

int main( void ) {
    cump_size_t a[N], b[N], c[N];
    cump_size_t *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(cump_size_t));
    hipMalloc( (void**)&dev_b, N * sizeof(cump_size_t));
    hipMalloc( (void**)&dev_c, N * sizeof(cump_size_t));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(cump_size_t),hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N * sizeof(cump_size_t),hipMemcpyHostToDevice);

    /**
     *  Aloca uma quantidade maior de blocos para o processamento dos dados
     *  Função add trata de não utilizar os dados desnecessários
     */
//    add<<<ceil((N+(TPB-1)/TPB)),TPB>>>(dev_a,dev_b,dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(cump_size_t),hipMemcpyDeviceToHost);

    // display the results
    for (int i=0; i<N; i++) {
	if(i%10==0)
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
